#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>

#include "../../hpp/skills.hpp"

#define TASK_NB 40
#define WARP_SIZE 32
#define LIST_SIZE 9 // 实际容量要减1
#define NEXT_TASK(ID) ((ID + 1) % LIST_SIZE)

/*任务结构体*/
struct Task
{
    // 任务编号
    int id;
    // 数据数量 [1,WARP_SIZE]
    int nb;
    // 数据地址（设备端）
    int *pDevData;
    // 处理结果（设备端）
    int *pDevResult;
    // 处理结果（主机端）
    int *pHostResult;
    // true表示结果可以被保存
    // false表示结果尚未得出
    bool canSave;
    Task()
    {
        id = 0;
        nb = 0;
        pDevData = NULL;
        pDevResult = NULL;
        pHostResult = NULL;
        canSave = false;
    }
};

/*任务列表（主机端）*/
static struct Task *list;
/*首尾任务在列表中的ID（主机端）*/
static int *flag;
/*任务完成数量（主机端）*/
static int *finTaksNb;

/*任务列表（设备端）*/
static struct Task *devList;
/*首尾任务在列表中的ID（设备端）*/
static int *devFlag;
/*任务完成数量（设备端）*/
static int *devFinTaksNb;

/*主机端->设备端内存拷贝流*/
static hipStream_t streamHd;

/*设备端->主机端内存拷贝流*/
static hipStream_t streamDh;

/*内核执行流*/
static hipStream_t streamKernel;

/*CUDA异常处理*/
static hipError_t err;

/*主机端生产者*/
void *cpuProducer(void *argc)
{
    for (int i = 1; i <= TASK_NB; i++)
    {
        bool temp = false;
        while (flag[0] == NEXT_TASK(flag[1]))
        {
            if (temp == false)
            {
                printf("[cpu] 队列是满的\n");
                temp = true;
            }
        }
        int cur = flag[1];
        list[cur].nb = rand() % (WARP_SIZE - 1 + 1) + 1;
        int bytes = sizeof(int) * list[cur].nb;
        int *data = (int *)malloc(bytes);
        for (int j = 0; j < list[cur].nb; j++)
        {
            data[j] = i;
        }
        err = hipMemcpyAsync(list[cur].pDevData, data, bytes, hipMemcpyHostToDevice, streamHd);
        if (err != 0)
        {
            printf("[hipError_t] hipMemcpyAsync返回0x%x\n", err);
            exit(1);
        }
        err = hipMemcpyAsync(list[cur].pDevResult, list[cur].pHostResult, bytes, hipMemcpyHostToDevice, streamHd);
        if (err != 0)
        {
            printf("[hipError_t] hipMemcpyAsync返回0x%x\n", err);
            exit(1);
        }
        hipStreamSynchronize(streamHd);
        list[cur].id = i;
        flag[1] = NEXT_TASK(cur);
        free(data);
        printf("[cpu] 在%d处插入任务%d\n", cur, i);
    }
    return NULL;
}

/*设备端消费者*/
__global__ void gpuConsumer(struct Task *devList, int *devFlag, int *devFinTaksNb)
{
    int threadId = threadIdx.x;
    while (devFinTaksNb[0] != TASK_NB)
    {
        bool temp = false;
        while (true)
        {
            // 稍微阻赛一会当前线程以确保之前对全局/共享内存的写入对其他线程可见
            // 这里用于获取全局内存中devFlag的最新值
            __threadfence();
            if (devFlag[0] != devFlag[1])
            {
                break;
            }
            if (threadId == 0)
            {
                if (temp == false)
                {
                    printf("[gpu] 队列是空的\n");
                    temp = true;
                }
            }
        }

        int cur = devFlag[0];
        if (threadId < devList[cur].nb)
        {
            int task = devList[cur].pDevData[threadId];
            devList[cur].pDevResult[threadId] = pow(task, 2) - task;
        }

        // 同步当前线程块中所有能够到达此处的线程
        __syncthreads();

        devList[cur].canSave = true;
        if (threadId == 0)
        {
            printf("[gpu] %d处的任务%d处理完成\n", cur, devList[cur].id);
        }
    }
}

/*保存结果到文件*/
void *cpuSaver(void *argc)
{
    while (finTaksNb[0] != TASK_NB)
    {
        int cur = flag[0];
        while (list[cur].canSave == false)
        {
        }
        int bytes = sizeof(int) * list[cur].nb;
        err = hipMemcpyAsync(list[cur].pHostResult, list[cur].pDevResult, bytes, hipMemcpyDeviceToHost, streamDh);
        if (err != 0)
        {
            printf("[hipError_t] hipMemcpyAsync返回0x%x\n", err);
            exit(1);
        }
        hipStreamSynchronize(streamDh);
        FILE *fp = fopen("result.txt", "a+");
        fprintf(fp, "%d\t", list[cur].id);
        for (int i = 0; i < list[cur].nb; i++)
        {
            fprintf(fp, "%d", list[cur].pHostResult[i]);
            if (i < list[cur].nb - 1)
            {
                fprintf(fp, " ");
            }
        }
        fprintf(fp, "\n");
        fclose(fp);
        flag[0] = NEXT_TASK(cur);
        printf("[cpu] %d处的任务%d结果已经保存\n", cur, list[cur].id);
        (finTaksNb[0])++;
        list[cur].canSave = false;
    }
    return NULL;
}

/*初始化*/
void init()
{
    remove("result.txt");
    int listBytes = LIST_SIZE * sizeof(struct Task);
    int flagBytes = 2 * sizeof(int);

    hipHostMalloc((void **)&list, listBytes, hipHostMallocMapped);
    hipHostMalloc((void **)&flag, flagBytes, hipHostMallocMapped);
    hipHostMalloc((void **)&finTaksNb, sizeof(int), hipHostMallocMapped);
    memset(flag, 0, flagBytes);
    memset(finTaksNb, 0, sizeof(int));

    for (int i = 0; i < LIST_SIZE; i++)
    {
        err = hipMalloc((void **)&(list[i].pDevData), sizeof(int) * WARP_SIZE);
        if (err != 0)
        {
            printf("[hipError_t] hipMalloc返回0x%x\n", err);
            exit(1);
        }
        err = hipMalloc((void **)&(list[i].pDevResult), sizeof(int) * WARP_SIZE);
        if (err != 0)
        {
            printf("[hipError_t] hipMalloc返回0x%x\n", err);
            exit(1);
        }
        list[i].pHostResult = (int *)malloc(sizeof(int) * WARP_SIZE);
        memset(list[i].pHostResult, 0, sizeof(int) * WARP_SIZE);
    }

    hipStreamCreate(&streamHd);
    hipStreamCreate(&streamDh);
    hipStreamCreate(&streamKernel);

    hipHostGetDevicePointer<struct Task>(&devList, (void *)list, 0);
    hipHostGetDevicePointer<int>(&devFlag, (void *)flag, 0);
    hipHostGetDevicePointer<int>(&devFinTaksNb, (void *)finTaksNb, 0);

    printf("初始化已完成\n");
}

/*清理*/
void free()
{
    hipStreamDestroy(streamHd);
    hipStreamDestroy(streamDh);
    hipStreamDestroy(streamKernel);

    for (int i = 0; i < LIST_SIZE; i++)
    {
        hipFree(list[i].pDevData);
        hipFree(list[i].pDevResult);
        free(list[i].pHostResult);
    }

    hipHostFree(list);
    hipHostFree(flag);
    hipHostFree(finTaksNb);
}

int main(int argc, char *argv[])
{
    char path[256];
    getWorkDir(path, sizeof(path), true);
    changeWorkDir(argv);

    init();

    pthread_t cpu_pro, cpu_sav;
    pthread_create(&cpu_sav, NULL, cpuSaver, NULL);
    gpuConsumer<<<1, WARP_SIZE, 0, streamKernel>>>(devList, devFlag, devFinTaksNb);
    pthread_create(&cpu_pro, NULL, cpuProducer, NULL);

    pthread_join(cpu_pro, NULL);
    printf("cpuProducer已经退出\n");
    hipDeviceSynchronize();
    printf("gpuConsumer已经退出\n");
    pthread_join(cpu_sav, NULL);
    printf("cpuSaver已经退出\n");

    free();
    return 0;
}
////////////////////////////////////////////////////////////////////////////////
// cd cuda/cpu_gpu_syn;make run;cd ../..
// cd cuda/cpu_gpu_syn;make clean;cd ../..
////////////////////////////////////////////////////////////////////////////////