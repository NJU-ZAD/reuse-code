#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "head.hpp"

__global__ void run()
{
    devKernel();
}

int main()
{
    int bytes = DATA_SIZE * sizeof(float);
    float data[DATA_SIZE];
    for (int i = 0; i < DATA_SIZE; i++)
    {
        data[i] = (i + 10) * 3.689;
        printf("START-%f\n", data[i]);
    }
    hipMemcpyToSymbol(HIP_SYMBOL(constData), &data, sizeof(data));
    hipMalloc((void **)&_data, bytes);
    hipMemcpy(_data, data, bytes, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(deviceData), &_data, sizeof(_data));

    run<<<1, DATA_SIZE>>>();
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), bytes);
    for (int i = 0; i < DATA_SIZE; i++)
    {
        printf("MID-%f\n", data[i]);
    }
    hipMemcpy(data, _data, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < DATA_SIZE; i++)
    {
        printf("END-%f\n", data[i]);
    }

    hipFree(_data);
    return 0;
}
////////////////////////////////////////////////////////////////////////////////
// cd cuda/usage;make run;cd ../..
// cd cuda/usage;make clean;cd ../..
////////////////////////////////////////////////////////////////////////////////